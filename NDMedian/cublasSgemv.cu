#include "hip/hip_runtime.h"
// nvcc 014 sgemv .c -lcublas
# include < stdio .h >
# include < stdlib .h >
# include < math .h >
# include < cuda_runtime .h >
# include " cublas_v2 .h"
# define IDX2C (i ,j , ld ) ((( j )*( ld ))+( i ))
# define m 6 // number of rows of a
# define n 5 // number of columns of a


__global__ void execute_gemv(hipblasHandle_t handle, float * d_a, float * d_x, float * d_y){
    float al = 1.0;
    float beta = 0.0;
    stat=hipblasSgemv(handle,HIPBLAS_OP_N,m,n,&al,d_a,m,d_x,1,&beta,d_y,1);
}

int main ( void ){
    hipError_t cudaStat ; // hipMalloc status
    hipblasStatus_t stat ; // CUBLAS functions status
    hipblasHandle_t handle ; // CUBLAS context

    int i,j; // i-row index , j- column index
    float * a; // a -mxn matrix on the host
    float * x; // x - n- vector on the host
    float * y; // y - m- vector on the host

    a=( float *) malloc (m*n* sizeof ( float )); // host mem. alloc for a
    x=( float *) malloc (n* sizeof ( float )); // host mem. alloc for x
    y=( float *) malloc (m* sizeof ( float )); // host mem. alloc for y

    // define an mxn matrix a - column by column
    int ind =11; // a:
    for(j=0;j<n;j ++){
        for(i=0;i<m;i ++){
            a[ IDX2C (i,j,m )]=( float )ind ++; // 13 ,19 ,25 ,31 ,37
        } 
    }

    printf ("a:\n");

    for (i=0;i<m;i ++){
        for (j=0;j<n;j ++){
            printf (" %4.0 f",a[ IDX2C (i,j,m )]); // print a row by row
        }
        printf ("\n");
    }

    for(i=0;i<n;i++) x[i ]=1.0 f; // x={1 ,1 ,1 ,1 ,1}^T
    for(i=0;i<m;i++) y[i ]=0.0 f; // y={0 ,0 ,0 ,0 ,0 ,0}^T

    // on the device
    float * d_a; // d_a - a on the device
    float * d_x; // d_x - x on the device
    float * d_y; // d_y - y on the device

    cudaStat = hipMalloc (( void **)& d_a ,m*n* sizeof (*a)); // device
    // memory alloc for a
    cudaStat = hipMalloc (( void **)& d_x ,n* sizeof (*x)); // device
    // memory alloc for x
    cudaStat = hipMalloc (( void **)& d_y ,m* sizeof (*y)); // device
    // memory alloc for y
    stat = hipblasCreate (& handle );
    stat = hipblasSetMatrix (m,n, sizeof (*a) ,a,m,d_a ,m); // cp a- >d_a
    stat = hipblasSetVector (n, sizeof (*x) ,x ,1 ,d_x ,1); // cp x- >d_x
    stat = hipblasSetVector (m, sizeof (*y) ,y ,1 ,d_y ,1); // cp y- >d_y
    float al =1.0 f; // al =1
    float bet =0.0 f; // bet =0
    // matrix - vector multiplication : d_y = al*d_a *d_x + bet *d_y
    // d_a - mxn matrix ; d_x - n-vector , d_y - m- vector ;
    // al ,bet - scalars

    execute_gemv <<<1,1>>> (handle, d_a, d_x, d_y);

    stat = hipblasGetVector (m, sizeof (*y) ,d_y ,1 ,y ,1); // copy d_y - >y
    printf ("y after Sgemv ::\ n");
    for(j=0;j<m;j ++){
        printf (" %5.0 f",y[j]); // print y after Sgemv
        printf ("\n");
    }

    hipFree (d_a ); // free device memory
    hipFree (d_x ); // free device memory
    hipFree (d_y ); // free device memory
    hipblasDestroy ( handle ); // destroy CUBLAS context
    free (a); // free host memory
    free (x); // free host memory
    free (y); // free host memory
    return EXIT_SUCCESS ;
}
